/* NAIVE_2D_CONVOLUTION*/

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef float DATA_TYPE;

__global__ void naive_kernel(DATA_TYPE *A, DATA_TYPE *B, const int problem_size)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;

	if ((i < problem_size-1) && (j < problem_size-1) && (i > 0) && (j > 0))
	{
		B[i * problem_size + j] =  c11 * A[(i - 1) * problem_size + (j - 1)] + 
							       c21 * A[(i - 1) * problem_size + (j + 0)] + 
								   c31 * A[(i - 1) * problem_size + (j + 1)] +
			 				       c12 * A[(i + 0) * problem_size + (j - 1)] + 
								   c22 * A[(i + 0) * problem_size + (j + 0)] +
								   c32 * A[(i + 0) * problem_size + (j + 1)] +
			 					   c13 * A[(i + 1) * problem_size + (j - 1)] + 
								   c23 * A[(i + 1) * problem_size + (j + 0)] +
								   c33 * A[(i + 1) * problem_size + (j + 1)];
	}
}

void naive_2D(DATA_TYPE* A, DATA_TYPE* B_dev, const int b_dim, const int mat_dim)
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

    hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim);

	if ((A_gpu == NULL) || (B_gpu == NULL))
		printf("allocation error on the device side\n");
	
	dim3 block(b_dim, b_dim, 1);
	dim3 grid(ceil(((float)mat_dim) / ((float)block.x)), ceil( ((float)mat_dim) / ((float)block.y)), 1);

	hipEvent_t start, stop;

    hipError_t err = hipEventCreate(&start);
	if (err != hipSuccess)
	{	
		printf("Error : %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}
	err = hipEventCreate(&stop);
	if (err != hipSuccess)
	{
		printf("Error : %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}
	err = hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * mat_dim * mat_dim, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("Error for hipMemcpy : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}
	hipEventRecord(start);
	naive_kernel<<<grid, block>>>(A_gpu, B_gpu, mat_dim);
	hipEventRecord(stop);

    err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error for hipMemcpy : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	err = hipMemcpy(B_dev, B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("Error for hipMemcpy : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("%0.4lf\n", milliseconds);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
}
