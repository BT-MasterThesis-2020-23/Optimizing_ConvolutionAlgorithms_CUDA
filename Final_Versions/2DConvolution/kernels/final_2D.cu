#include "hip/hip_runtime.h"
/* CONSTANT_MEM_H */

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef float DATA_TYPE;

__constant__ DATA_TYPE c_3[3][3] = {{0.2, -0.3, 0.4}, {0.5, 0.6, 0.7}, {-0.8, -0.9, 0.1}};

/*
	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;
	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;
*/

__global__ void final_version_kernel(DATA_TYPE *A, DATA_TYPE *B, 
                            const int mat_dim, const int b_dim)
{
	__shared__ float shmem[34][34];

    short int gl_ty = blockIdx.x * blockDim.x + threadIdx.x;
    short int gl_tx = blockIdx.y * blockDim.y + threadIdx.y;

    short int lcl_ty = threadIdx.x;
    short int lcl_tx = threadIdx.y;

    if ((gl_ty > 0) && (gl_tx > 0) && (gl_tx < mat_dim -1) && (gl_ty < mat_dim -1))
    {
        shmem[lcl_tx + 1][lcl_ty + 1] = A[gl_tx * mat_dim + gl_ty];

        if(lcl_ty == 0)
            shmem[lcl_tx + 1][0] = A[gl_tx * mat_dim + gl_ty - 1];

        if(lcl_tx == 0)
            shmem[0][lcl_ty + 1] = A[(gl_tx - 1) * mat_dim + gl_ty];

        if(lcl_ty == (b_dim - 1))
            shmem[lcl_tx + 1][b_dim + 1] = A[gl_tx * mat_dim + gl_ty + 1];

        if(lcl_tx == (b_dim - 1))
            shmem[b_dim + 1][lcl_ty + 1] = A[(gl_tx + 1) * mat_dim + gl_ty];
        __syncthreads();

        B[gl_tx * mat_dim + gl_ty] = c_3[0][0] * shmem[lcl_tx][lcl_ty] +
                                    c_3[1][0] * shmem[lcl_tx][lcl_ty + 1] + 
                                    c_3[2][0] * shmem[lcl_tx][lcl_ty + 2] + 
                                    c_3[0][1] * shmem[lcl_tx + 1][lcl_ty ] + 
                                    c_3[1][1] * shmem[lcl_tx + 1][lcl_ty + 1] + 
                                    c_3[2][1] * shmem[lcl_tx + 1][lcl_ty + 2] + 
                                    c_3[0][2] * shmem[lcl_tx + 2][lcl_ty ] + 
                                    c_3[1][2] * shmem[lcl_tx + 2][lcl_ty + 1] + 
                                    c_3[2][2] * shmem[lcl_tx + 2][lcl_ty + 2];
    }
}


void final_2D(DATA_TYPE* A, DATA_TYPE* B_dev, const int b_dim, 
                  const int mat_dim, const int num_streams)
{
    dim3 threadsPerBlock(b_dim, b_dim, 1);
    dim3 blocksPerGrid(ceil(mat_dim/b_dim), ceil(mat_dim/(b_dim * num_streams)), 1);

   	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

    hipError_t err = hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

	err = hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

    hipStream_t streams[num_streams];
    const int chunk_size = ceil((mat_dim * mat_dim)/num_streams);
	float milliseconds = 0;

    for(int i = 0; i < num_streams; i++)
    {
        hipStreamCreate(&streams[i]);
    }

    for(int stream = 0; stream < num_streams; stream++) 
    {
        const int lower = chunk_size * stream;      
        const int upper = min(lower + chunk_size, mat_dim * mat_dim);
        const int width = upper - lower;

        hipEvent_t start, stop;
        float x = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMemcpyAsync(A_gpu + lower, A + lower, sizeof(DATA_TYPE) * width, hipMemcpyHostToDevice, streams[stream]);
		hipEventRecord(start);
    	final_version_kernel<<<blocksPerGrid, threadsPerBlock, 0, streams[stream]>>>(A_gpu + lower, 
                                                                        B_gpu + lower, mat_dim, num_streams);
		hipEventRecord(stop);
        hipMemcpyAsync(B_dev + lower, B_gpu + lower, sizeof(DATA_TYPE) * width, hipMemcpyDeviceToHost, streams[stream]);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&x, start, stop);

        milliseconds += x;

        if(stream)  
            hipStreamSynchronize(streams[stream-1]);
    }
    for(int i = 0; i < num_streams; i++)
    {
        hipStreamDestroy(streams[i]);
    }
	printf("%0.4lf\n", milliseconds);

	hipFree(A_gpu);
	hipFree(B_gpu);    
}