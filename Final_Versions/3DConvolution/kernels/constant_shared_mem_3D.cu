/* CONSTANT_SHARED_MEM_3D_H */

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef float DATA_TYPE;

__constant__ DATA_TYPE c_cs[3][3] = {{2, -3, 4}, {5, 6, 7}, {-8, -9, 10}};

__global__ void const_shared_mem_conv3D_kernel(DATA_TYPE *A, DATA_TYPE *B, int mat_dim, 
                        const short int b_dim, const short int i)
{
	__shared__ float shmem[3][34][34];

    short int gl_ty = blockIdx.x * blockDim.x + threadIdx.x;
    short int gl_tx = blockIdx.y * blockDim.y + threadIdx.y;

    short int lcl_ty = threadIdx.x;
    short int lcl_tx = threadIdx.y;

    if ((gl_ty > 0) && (gl_tx > 0) && (gl_tx < mat_dim -1) && (gl_ty < mat_dim -1) &&
        (i > 0) && (i < mat_dim -1))
    {
        shmem[0][lcl_tx+1][lcl_ty+1] = A[(i-1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty];
        shmem[1][lcl_tx+1][lcl_ty+1] = A[i * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty];
        shmem[2][lcl_tx+1][lcl_ty+1] = A[(i+1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty];

        if(lcl_ty == 0)
        {
            shmem[0][lcl_tx + 1][0] = A[(i-1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty - 1];
            shmem[2][lcl_tx + 1][0] = A[(i+1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty - 1];
            shmem[1][lcl_tx + 1][0] = A[i * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty - 1];
        }

        if(lcl_tx == 0)
        {
            shmem[0][0][lcl_ty + 1] = A[(i-1) * mat_dim * mat_dim + (gl_tx - 1) * mat_dim + gl_ty];
            shmem[2][0][lcl_ty + 1] = A[(i+1) * mat_dim * mat_dim + (gl_tx - 1) * mat_dim + gl_ty];
            shmem[1][0][lcl_ty + 1] = A[i * mat_dim * mat_dim + (gl_tx - 1) * mat_dim + gl_ty];
        }

        if(lcl_ty == (b_dim - 1))
        {
            shmem[0][lcl_tx + 1][b_dim + 1] = A[(i-1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty + 1];
            shmem[2][lcl_tx + 1][b_dim + 1] = A[(i+1) * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty + 1];
            shmem[1][lcl_tx + 1][b_dim + 1] = A[i * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty + 1];
        }

        if(lcl_tx == (b_dim - 1))
        {
            shmem[0][b_dim + 1][lcl_ty + 1] = A[(i-1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty];
            shmem[2][b_dim + 1][lcl_ty + 1] = A[(i+1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty];
            shmem[1][b_dim + 1][lcl_ty + 1] = A[i * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty];
        }

        if ((lcl_tx == (b_dim -1)) && (lcl_ty == (b_dim -1)))
        {
            shmem[0][b_dim + 1][b_dim + 1] = A[(i-1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty + 1];
            shmem[2][b_dim + 1][b_dim + 1] = A[(i+1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty + 1];
            shmem[1][b_dim + 1][b_dim + 1] = A[i * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty + 1];
        }

        if ((lcl_tx == 0) && (lcl_ty == (b_dim -1)))
        {
            shmem[0][0][b_dim + 1] = A[(i-1) * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty + 1];
            shmem[2][0][b_dim + 1] = A[(i+1) * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty + 1];
            shmem[1][0][b_dim + 1] = A[i * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty + 1];
        }

        if ((lcl_ty == 0) && (lcl_tx == (b_dim -1)))
        {
            shmem[0][b_dim + 1][0] = A[(i-1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty - 1];
            shmem[2][b_dim + 1][0] = A[(i+1) * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty - 1];
            shmem[1][b_dim + 1][0] = A[i * mat_dim * mat_dim + (gl_tx + 1) 
                                                * mat_dim + gl_ty - 1];
        }

        if ((lcl_ty == 0) && (lcl_tx == 0))
        {
            shmem[0][0][0] = A[(i-1) * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty - 1];
            shmem[2][0][0] = A[(i+1) * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty - 1];
            shmem[1][0][0] = A[i * mat_dim * mat_dim + (gl_tx - 1) 
                                                * mat_dim + gl_ty - 1];
        }
        __syncthreads();

        B[i * mat_dim * mat_dim + gl_tx * mat_dim + gl_ty] = 
                                 c_cs[0][0] * (shmem[0][lcl_tx][lcl_ty] + shmem[0][lcl_tx][lcl_ty+2]) +
                                 c_cs[1][0] * (shmem[0][lcl_tx+1][lcl_ty] + shmem[0][lcl_tx+1][lcl_ty+2]) + 
                                 c_cs[2][0] * (shmem[0][lcl_tx+2][lcl_ty] + shmem[0][lcl_tx+2][lcl_ty+2]) + 
                                 c_cs[0][1] * shmem[1][lcl_tx][lcl_ty+1] + 
                                 c_cs[1][1] * shmem[1][lcl_tx+1][lcl_ty+1] + 
                                 c_cs[2][1] * shmem[1][lcl_tx+2][lcl_ty+1] + 
                                 c_cs[0][2] * (shmem[2][lcl_tx][lcl_ty] + shmem[2][lcl_tx][lcl_ty+2]) + 
                                 c_cs[1][2] * (shmem[2][lcl_tx+1][lcl_ty] + shmem[2][lcl_tx+1][lcl_ty+2]) + 
                                 c_cs[2][2] * (shmem[2][lcl_tx+2][lcl_ty] + shmem[2][lcl_tx+2][lcl_ty+2]);
    }
}


void constant_shared_mem_3D(DATA_TYPE* A, DATA_TYPE* B_dev, const int b_dim, int mat_dim)
{
    dim3 threadsPerBlock((size_t)b_dim, (size_t)b_dim, 1);
    dim3 blocksPerGrid((size_t)ceil(mat_dim/b_dim), size_t(ceil(mat_dim/b_dim)), 1);
        
   	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

    hipError_t err = hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

	err = hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

   	hipEvent_t start, stop;
    err = hipEventCreate(&start);
	if (err != hipSuccess)
	{
		printf("Error : %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	err = hipEventCreate(&stop);
	if (err != hipSuccess)
	{
		printf("Error : %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	err = hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim, 
                    hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
        printf("A\n");
		printf("Error for hipMemcpy : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	hipEventRecord(start);
    for (int i = 1; i < mat_dim - 1; i++)
    {
    	const_shared_mem_conv3D_kernel<<<blocksPerGrid, threadsPerBlock>>>(A_gpu, B_gpu, mat_dim, b_dim, i);
        hipDeviceSynchronize();
    }
	hipEventRecord(stop);

    err = hipGetLastError();
	if (err != hipSuccess)
	{
        printf("Kernel\n");
		printf("Error for kernel : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	err = hipMemcpy(B_dev, B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim, 
                    hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
        printf("B\n");
		printf("Error for hipMemcpy : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);       
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%0.4lf\n", milliseconds);	
	hipFree(A_gpu);
	hipFree(B_gpu);
}


