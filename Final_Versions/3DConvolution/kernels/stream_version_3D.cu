#include "hip/hip_runtime.h"
/*STREAM_VERSION_3D_CONVOLUTION*/

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef float DATA_TYPE;

__global__ void stream_conv3D_kernel(DATA_TYPE *A, DATA_TYPE *B, const int mat_dim, 
                                     const short i)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;

	if ((i < (mat_dim-1)) && (j < (mat_dim-1)) &&  (k < (mat_dim-1)) && (i > 0) && (j > 0) && (k > 0))
	{
		B[i*(mat_dim * mat_dim) + j*mat_dim + k] = 
                c11 * (A[(i - 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k - 1)] +  
          				A[(i - 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 1)]) +
			    c13 * (A[(i + 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k - 1)] +
		                A[(i + 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 1)]) +
				c21 * (A[(i - 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k - 1)] +
   						A[(i - 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 1)]) +
				c23 * (A[(i + 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k - 1)] + 
        				A[(i + 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 1)]) + 
				c31 * (A[(i - 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k - 1)] +
		        		A[(i - 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 1)]) +
				c33 * (A[(i + 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k - 1)] + 
		        		A[(i + 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 1)]) +
    			c12 * A[(i + 0)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 0)] + 
				c22 * A[(i + 0)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 0)] +
				c32 * A[(i + 0)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 0)] ;
	}
}

void stream_version_3D(DATA_TYPE* A, DATA_TYPE* B_dev, const int b_dim, 
                        const int mat_dim, const int num_streams)
{
    if (num_streams >= 32)
    {
        printf("please enter stream number below than 32\n");
        exit(1);
    }

    dim3 threadsPerBlock(b_dim, b_dim, 1);
    dim3 blocksPerGrid(size_t(mat_dim/b_dim), size_t(mat_dim/b_dim),  1);

   	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

    hipError_t err = hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * mat_dim
                                 * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

	err = hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }
    float milliseconds = 0;
    
    hipStream_t streams[num_streams];

    const int chunk_size = ceil((mat_dim * mat_dim * mat_dim)/num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        err = hipStreamCreate(&streams[i]);
        if (err != hipSuccess)
        {
            printf("Error for hipStreamCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
    }

    for(int stream = 0; stream < num_streams; stream++) 
    {
        const int lower = chunk_size * stream;
        const int upper = min(lower + chunk_size, mat_dim * mat_dim * mat_dim);
        const int width = upper - lower;
        hipEvent_t start, stop;
        float x = 0;
        err = hipEventCreate(&start);
        if (err != hipSuccess)
        {
            printf("Error for hipEventCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        err = hipEventCreate(&stop);
        if (err != hipSuccess)
        {
            printf("Error for hipEventCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        if ((stream != 0) && (stream != num_streams - 1))
            err = hipMemcpyAsync(A_gpu + lower - mat_dim * mat_dim,
                                  A + lower - mat_dim * mat_dim,
                                  sizeof(DATA_TYPE) * (width + 2 * mat_dim * mat_dim),
                                  hipMemcpyHostToDevice, streams[stream]);
        else if (stream == 0)
            err = hipMemcpyAsync(A_gpu, 
                                  A, 
                                  sizeof(DATA_TYPE) * width + mat_dim * mat_dim, 
                                  hipMemcpyHostToDevice, streams[stream]);

        else if (stream == num_streams - 1)
            err = hipMemcpyAsync(A_gpu + lower, 
                                  A + lower, 
                                  sizeof(DATA_TYPE) * width, 
                                  hipMemcpyHostToDevice, streams[stream]);

        if (err != hipSuccess)
        {
            printf("A\n");
            printf("Error for hipMemcpyAsync : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        for (int j = 0; j < mat_dim/num_streams; j++)
        {   
            x = 0;
            hipEventRecord(start);
            stream_conv3D_kernel<<<blocksPerGrid, threadsPerBlock, 0, streams[stream]>>>
                            (A_gpu, B_gpu, mat_dim, j + 
                            (stream * (mat_dim/num_streams)));
            hipEventRecord(stop);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("Kernel\n");
                printf("Error for kernel : %s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);       
            }
            hipEventSynchronize(stop);
            hipEventElapsedTime(&x, start, stop);
            milliseconds += x;
        }
        err = hipMemcpyAsync(B_dev + lower, 
                              B_gpu + lower, 
                              sizeof(DATA_TYPE) * width, 
                              hipMemcpyDeviceToHost, 
                              streams[stream]);
        if (err != hipSuccess)
        {
            printf("B\n");
            printf("Error for hipMemcpyAsync : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }

        if(stream)  
            hipStreamSynchronize(streams[stream-1]);
    }
    for(int i = 0; i < num_streams; i++)
    {
        hipStreamDestroy(streams[i]);
    }
    printf("%0.4lf\n", milliseconds);
	hipFree(A_gpu);
	hipFree(B_gpu);    
}