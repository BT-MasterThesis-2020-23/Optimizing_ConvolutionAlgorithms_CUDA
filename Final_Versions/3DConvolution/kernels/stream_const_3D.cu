#include "hip/hip_runtime.h"
/*STREAM_AND_CONSTANT_MEM_3D_CONVOLUTION*/

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef float DATA_TYPE;

__constant__ DATA_TYPE c_s[3][3] = {{2, -3, 4}, {5, 6, 7}, {-8, -9, 10}};

__global__ void stream_const_conv3D_kernel(DATA_TYPE *A, DATA_TYPE *B, const int mat_dim, 
                                           const short i)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < (mat_dim-1)) && (j < (mat_dim-1)) &&  (k < (mat_dim-1)) && (i > 0) && (j > 0) && (k > 0))
	{
		B[i*(mat_dim * mat_dim) + j*mat_dim + k] = 
                c_s[0][0] * (A[(i - 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k - 1)] +  
          				A[(i - 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 1)]) +
			    c_s[0][2] * (A[(i + 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k - 1)] +
		                A[(i + 1)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 1)]) +
				c_s[1][0] * (A[(i - 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k - 1)] +
   						A[(i - 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 1)]) +
				c_s[1][2] * (A[(i + 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k - 1)] + 
        				A[(i + 1)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 1)]) + 
				c_s[2][0] * (A[(i - 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k - 1)] +
		        		A[(i - 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 1)]) +
				c_s[2][2] * (A[(i + 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k - 1)] + 
		        		A[(i + 1)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 1)]) +
    			c_s[0][1] * A[(i + 0)*(mat_dim * mat_dim) + (j - 1)*mat_dim + (k + 0)] + 
				c_s[1][1] * A[(i + 0)*(mat_dim * mat_dim) + (j + 0)*mat_dim + (k + 0)] +
				c_s[2][1] * A[(i + 0)*(mat_dim * mat_dim) + (j + 1)*mat_dim + (k + 0)] ;
	}
}

void stream_const_3D(DATA_TYPE* A, DATA_TYPE* B_dev, const int b_dim, 
                        const int mat_dim, const int num_streams)
{
    if (num_streams >= 32)
    {
        printf("please enter stream number below than 32\n");
        exit(1);
    }

    dim3 threadsPerBlock(b_dim, b_dim, 1);
    dim3 blocksPerGrid(size_t(mat_dim/b_dim), size_t(mat_dim/b_dim),  1);

   	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

    hipError_t err = hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * mat_dim
                                 * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }

	err = hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim);
    if(err != hipSuccess)
    {
        printf("unsuccessful cuda malloc operation for stream_const_mem_2D\n");
        exit(EXIT_FAILURE);       
    }
    float milliseconds = 0;
    
    hipStream_t streams[num_streams];

    const int chunk_size = ceil((mat_dim * mat_dim * mat_dim)/num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        err = hipStreamCreate(&streams[i]);
        if (err != hipSuccess)
        {
            printf("Error for hipStreamCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
    }

    for(int stream = 0; stream < num_streams; stream++) 
    {
        const int lower = chunk_size * stream;
        const int upper = min(lower + chunk_size, mat_dim * mat_dim * mat_dim);
        const int width = upper - lower;
        hipEvent_t start, stop;
        float x = 0;
        err = hipEventCreate(&start);
        if (err != hipSuccess)
        {
            printf("Error for hipEventCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        err = hipEventCreate(&stop);
        if (err != hipSuccess)
        {
            printf("Error for hipEventCreate : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        if ((stream != 0) && (stream != num_streams - 1))
            err = hipMemcpyAsync(A_gpu + lower - mat_dim * mat_dim,
                                  A + lower - mat_dim * mat_dim,
                                  sizeof(DATA_TYPE) * (width + 2 * mat_dim * mat_dim),
                                  hipMemcpyHostToDevice, streams[stream]);
        else if (stream == 0)
            err = hipMemcpyAsync(A_gpu, 
                                  A, 
                                  sizeof(DATA_TYPE) * width + mat_dim * mat_dim, 
                                  hipMemcpyHostToDevice, streams[stream]);

        else if (stream == num_streams - 1)
            err = hipMemcpyAsync(A_gpu + lower, 
                                  A + lower, 
                                  sizeof(DATA_TYPE) * width, 
                                  hipMemcpyHostToDevice, streams[stream]);

        if (err != hipSuccess)
        {
            printf("A\n");
            printf("Error for hipMemcpyAsync : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }
        for (int j = 0; j < mat_dim/num_streams; j++)
        {   
            x = 0;
            hipEventRecord(start);
            stream_const_conv3D_kernel<<<blocksPerGrid, threadsPerBlock, 0, streams[stream]>>>
                            (A_gpu, B_gpu, mat_dim, j + 
                            (stream * (mat_dim/num_streams)));
            hipEventRecord(stop);
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("Kernel\n");
                printf("Error for kernel : %s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);       
            }
            hipEventSynchronize(stop);
            hipEventElapsedTime(&x, start, stop);
            milliseconds += x;
        }
        err = hipMemcpyAsync(B_dev + lower, 
                              B_gpu + lower, 
                              sizeof(DATA_TYPE) * width, 
                              hipMemcpyDeviceToHost, 
                              streams[stream]);
        if (err != hipSuccess)
        {
            printf("B\n");
            printf("Error for hipMemcpyAsync : %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);       
        }

        if(stream)  
            hipStreamSynchronize(streams[stream-1]);
    }
    for(int i = 0; i < num_streams; i++)
    {
        hipStreamDestroy(streams[i]);
    }
    printf("%0.4lf\n", milliseconds);
	hipFree(A_gpu);
	hipFree(B_gpu);    
}