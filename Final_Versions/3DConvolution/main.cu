/**
 * 3DConvolution.cu: This file is optimized by burak topcu.
 * 
 * Contact: topcuuburak@gmail.com>
 * Teaching / Research Assistant at IZTECH
 * Web address: https://ceng.iyte.edu.tr/people/burak-topcu/
 * 
 * nvcc main.cu -o res
 * 1st argument is experiment type (--naive, --constant_mem, --stream_version etc.)
 * 2nd argument is matrix dimensions (1 dim of square matrix, 64, 128, 256 etc)
 * 3rd argument is thread block dimension size (8, 16, 32)
 * 4th argument is num_streams (2,4,8,16,32)
 * 5th argument cpu time (--cpu_time)
 * 
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "kernels/polybenchUtilFuncts.h"
#include "kernels/constant_mem_3D.cu"
#include "kernels/constant_shared_mem_3D.cu"
#include "kernels/final_3D.cu"
#include "kernels/naive_3D.cu"
#include "kernels/shared_mem_3D.cu"
#include "kernels/stream_const_3D.cu"
#include "kernels/stream_version_3D.cu"
#include "kernels/timer.cuh"

#define PERCENT_DIFF_ERROR_THRESHOLD 0.05
#define GPU_DEVICE 0

typedef float DATA_TYPE;

void conv3D(DATA_TYPE* A, DATA_TYPE* B, const int mat_dim)
{
	int i, j, k;

	const int NJ = mat_dim;
	const int NK = mat_dim;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;

	for (i = 1; i < mat_dim - 1; ++i) // 0
	{
		for (j = 1; j < NJ - 1; ++j) // 1
		{
			for (k = 1; k < NK -1; ++k) // 2
			{
				B[i*(NK * NJ) + j*NK + k] = c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)] +  
									c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k + 1)] +
								    c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)] +
									c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k + 1)] +
									c21 * A[(i - 1)*(NK * NJ) + (j + 0)*NK + (k - 1)] +
									c21 * A[(i - 1)*(NK * NJ) + (j + 0)*NK + (k + 1)] +
									c23 * A[(i + 1)*(NK * NJ) + (j + 0)*NK + (k - 1)] + 
									c23 * A[(i + 1)*(NK * NJ) + (j + 0)*NK + (k + 1)] +
									c31 * A[(i - 1)*(NK * NJ) + (j + 1)*NK + (k - 1)] +
									c31 * A[(i - 1)*(NK * NJ) + (j + 1)*NK + (k + 1)] +
									c33 * A[(i + 1)*(NK * NJ) + (j + 1)*NK + (k - 1)] + 
									c33 * A[(i + 1)*(NK * NJ) + (j + 1)*NK + (k + 1)] +
									c12 * A[(i + 0)*(NK * NJ) + (j - 1)*NK + (k + 0)] + 
									c22 * A[(i + 0)*(NK * NJ) + (j + 0)*NK + (k + 0)] +
									c32 * A[(i + 0)*(NK * NJ) + (j + 1)*NK + (k + 0)] ;
			}
		}
	}
}

void init(DATA_TYPE* A, const int mat_dim)
{
	int i, j, k;

	const int NJ = mat_dim;
	const int NK = mat_dim;

	for (i = 0; i < mat_dim; ++i)
   	{
		for (j = 0; j < NJ; ++j)
		{
			for (k = 0; k < NK; ++k)
			{
				A[i*(NK * NJ) + j*NK + k] = (float)rand()/RAND_MAX;
			}
		}
	}
}


void compareResults(DATA_TYPE* B, DATA_TYPE* B_Gpu, const int mat_dim)
{
	const int NJ = mat_dim;
	const int NK = mat_dim;	

	int i, j, k, fail;
	fail = 0;

	for (i = 1; i < mat_dim - 1; ++i) 
	{
		for (j = 1; j < NJ - 1; ++j) 
		{
			for (k = 1; k < NK - 1; ++k) 
			{
				if (percentDiff(B[i*(NK * NJ) + j*NK + k], B_Gpu[i*(NK * NJ) + j*NK + k]) 
								> PERCENT_DIFF_ERROR_THRESHOLD)
				{
					fail++;
				}
			}	
		}
	}
	printf("Failure in percent: %0.4f\n",
		(float)((float)100*((float)fail))/((float)(mat_dim * mat_dim * mat_dim)));
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
    printf("Max Threads Per block %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max Threads Dim (x, y, z) = (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Max Threads Grid Size (x, y, z) = (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	hipSetDevice( GPU_DEVICE );
}

int main(int argc, char *argv[])
{	
	int block_size, mat_dim, num_streams;

	Timer timer;
	if (argv[2] != NULL)
		mat_dim = atoi(argv[2]);
	
	if (argv[3] != NULL)
		block_size = atoi(argv[3]);
	
	if (argv[4] != NULL)
		num_streams = atoi(argv[4]);

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* B_dev;

    hipHostMalloc((void **)&A, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim); 
    hipHostMalloc((void **)&B_dev, sizeof(DATA_TYPE) * mat_dim * mat_dim * mat_dim); 

	B = (DATA_TYPE*)malloc(mat_dim * mat_dim * mat_dim * sizeof(DATA_TYPE));

	if ((A == NULL) || (B == NULL) || (B_dev == NULL))
	{
		printf("Allocation error on the host side\n");
	}

	init(A, mat_dim);

	if (argv[1] != NULL)
	{
		printf("Matrix dim = %dx%dx%d, block_size = %dx%d, streams = %d\n", 
		mat_dim, mat_dim, mat_dim, block_size, block_size, num_streams);
        timer.start();
		if(strcmp(argv[1], "--naive") == 0)
		{
			printf("Naive version\n");
			naive_3D(A, B_dev, block_size, mat_dim);
		}
		if(strcmp(argv[1], "--constant_mem") == 0)
		{
			printf("With constant memory\n");
			constant_mem_3D(A, B_dev, block_size, mat_dim);
		}
		if(strcmp(argv[1], "--constant_shared_mem") == 0)
		{
			printf("With constant and shared memory\n");
			constant_shared_mem_3D(A, B_dev, block_size, mat_dim);
		}
		if(strcmp(argv[1], "--shared_mem") == 0)
		{	
			printf("With shared memory\n");
			shared_mem_3D(A, B_dev, block_size, mat_dim);
		}
		if(strcmp(argv[1], "--stream_version") == 0)
		{	
			printf("With stream implemented\n");
			stream_version_3D(A, B_dev, block_size, mat_dim, num_streams);
		}
		if(strcmp(argv[1], "--stream_const") == 0)
		{
			printf("With stream and constant memory implementation\n");
			stream_const_3D(A, B_dev, block_size, mat_dim, num_streams);
		}
		if(strcmp(argv[1], "--final") == 0)
		{
			printf("Final version\n");
			final_version_3D(A, B_dev, block_size, mat_dim, num_streams);
		}
		timer.stop("");
	}

	if (argv[5] != NULL)
	{
		if (strcmp(argv[5], "--cpu_time") == 0)
		{
			double t_start, t_end;

			t_start = rtclock();
			conv3D(A, B, mat_dim);
			t_end = rtclock();
			fprintf(stdout, "%0.6lf\n", t_end - t_start);

			compareResults(B, B_dev, mat_dim);
		}
	}

    hipHostFree(A);    
    hipHostFree(B_dev);
	free(B);
	
	return 0;
}




